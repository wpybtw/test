#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include <omp.h>
#include <assert.h>

#include "timer.h"

const size_t m = 10ull * 1000 * 1000;
const int n = 64;

__global__ void kernel(float *data, float *output)
{

    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    size_t ltid = threadIdx.x;
    size_t offset = blockDim.x * blockIdx.x;
    __shared__ float tmp[n][n];
    float tmax = -1000;

    if (tid < m)
    {
        for (int i = 0; i < n; i++)
        {
            tmp[i][ltid] = data[(offset + i) * n + ltid];
        }
        __syncthreads();

        for (int i = 0; i < n; i++)
        {
            tmax = fmax(tmp[ltid][(i + ltid) % n], tmax);
            // tmax = fmax(tmp[ltid][i], tmax);
        }
        output[tid] = tmax;
    }
}

void check(float *a, float *b)
{

#pragma omp parallel for
    for (int i = 0; i < m; i++)
    {
        float acc = -100;
        for (size_t j = 0; j < n; j++)
        {
            acc = fmax(acc, a[i * n + j]);
        }

        if (abs(acc - b[i]) > 0.001f)
        {
            std::cout << i << " " << acc << " " << b[i] << std::endl;
            // break;
        }
    }
    std::cout<<" verified "<<std::endl;
}

int main()
{
    std::srand(0);
    using T = float;
    std::vector<T> vA;
    std::vector<T> vB;

    size_t size_a = m * n * sizeof(T);
    size_t size_b = m * sizeof(T);

    std::cout << "using m=" << m << " n=" << n << std::endl;

    vA.resize(m * n);
    vB.resize(m);

    T *a, *b;
    hipMalloc(&a, size_a);
    hipMalloc(&b, size_b);

    std::generate(vA.begin(), vA.end(), std::rand);

    hipMemcpy(a, vA.data(), size_a, hipMemcpyHostToDevice);
    hipMemcpy(b, vB.data(), size_b, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    {
        Stopwatch t;
        kernel<<<m / n + 1, n>>>(a, b);
        hipDeviceSynchronize();
        int ms = t.Finish();
        std::cout << " CUDA kernel takes " << ms << " ms" << std::endl;
    }

    hipMemcpy(vB.data(), b, size_b, hipMemcpyDeviceToHost);

    check(vA.data(), vB.data());

    return 0;
}